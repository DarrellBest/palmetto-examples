#include <stdio.h>
#include <hip/hip_runtime.h>

#include <assert.h>

__global__ void add(int *a, int *b, int *c) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
    __syncthreads();
}


void random_ints(int* a, int N) {
    for (int i=0; i<N; i++){
        a[i] = rand() % 1000;
    }
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main(){
    int *a, *b, *c;
    int *a_d, *b_d, *c_d;
    int size = sizeof(int)*N;

    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);

    // setup initial values:
    a = (int*)malloc(size); random_ints(a, N);
    b = (int*)malloc(size); random_ints(b, N);
    c = (int*)malloc(size);

    hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);

    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(a_d, b_d, c_d);
    hipMemcpy(c, c_d, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        assert(a[i] + b[i] == c[i]);

    free(a); free(b); free(c);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}
